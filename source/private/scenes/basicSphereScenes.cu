#include "hip/hip_runtime.h"
#include "../../public/scenes/basicSphereScenes.h"

#include "hiprand/hiprand_kernel.h"
#include ""

#include "../../public/shapes/sphere.h"
#include "../../public/materials/dielectric.h"
#include "../../public/materials/metal.h"
#include "../../public/camera.h"
#include "../../public/util.h"
#include "../../public/bvh/bvh.h"

__global__ void initializeBasicSpheres(Shape** d_shapeList, bvhNode* d_bvhTree, Camera** d_camera, int pX, int pY, int objectCount, hiprandState* localCurandState)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        d_shapeList[0] = new Sphere(vec3(0.0f, 1.0f, 2.0f), 1.2f, new lambertian(color4(0.1f, 0.2f, 0.5f, 1.0f)));

        d_shapeList[1] = new Sphere(vec3(0.0f, -20.0f, 5.0f), 20.0f, new lambertian(color4(0.8f, 0.8f, 0.0f, 1.0f)));

        d_shapeList[2] = new Sphere(vec3(2.0f, 0.5f, 2.0f), 0.75f, new metal(color4(0.69f, 0.55f, 0.34f, 1.0f), 0.8f));

        d_shapeList[3] = new Sphere(vec3(-2.0f, 0.5f, 2.0f), 0.75f, new metal(color4(0.8f, 0.6f, 0.2f, 1.0f), 0.1f));

        /*
         * Hollow glass sphere (glass sphere with glass refractionIndex and air sphere)
         */
        d_shapeList[4] = new Sphere(vec3(1.0f, 0.5f, 0.5f), 0.5f, new dielectric(1.50f));

        d_shapeList[5] = new Sphere(vec3(1.0f, 0.5f, 0.5f), 0.40f, new dielectric(1.00f / 1.50f));

        *d_bvhTree = bvhNode(d_shapeList, objectCount);

        *d_camera = new Camera(vec3(0.0f, 1.5f, -3.0f), vec3(0.0f, 1.0f, 0.0f), vec2(-5.0f, 90.0f), 45.0f, pX, pY, AAMethod::MSAA1000, 5.0f, 0.0f); // standard camera
    }
}

void basicSphereScene::createScene(bvhNode*& d_bvhTree, Shape** d_shapeList, Camera** d_camera, int pX, int pY, hiprandState* localCurandState)
{
    initializeTree(objectCount, d_bvhTree);

    initializeBasicSpheres<<<1, 1>>>(d_shapeList, d_bvhTree, d_camera, pX, pY, objectCount, localCurandState);
}