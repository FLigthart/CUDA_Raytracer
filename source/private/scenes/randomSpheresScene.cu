#include "hip/hip_runtime.h"
#include "../../public/scenes/randomSpheresScene.h"

#include "hiprand/hiprand_kernel.h"
#include ""

#include "../../public/shapes/sphere.h"
#include "../../public/materials/dielectric.h"
#include "../../public/materials/metal.h"
#include "../../public/camera.h"
#include "../../public/bvh/bvh.h"
#include "../../public/util.h"

__global__ void initializeRandomScene(Shape** d_shapeList, bvhNode* d_bvhTree, Camera* d_camera, int pX, int pY, int objectCount, int randomSeed)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        INIT_RAND_LOCAL();

        material* groundMaterial = new lambertian(color4(0.5f, 0.5f, 0.5f, 1.0f));

        d_shapeList[0] = new Sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f, groundMaterial);

        int i = 1;
        for (int a = -11; a < 11; a++)
        {
            for (int b = -11; b < 11; b++)
            {
                float chosenMaterial = RND;
                vec3 center(static_cast<float>(a) + RND * 0.5f, 0.2f, static_cast<float>(b) + RND * 0.5f);
                if (chosenMaterial < 0.8f)
                {
                    vec3 centerTwo = center + vec3(0.0f, 0.25f * hiprand_uniform(&localRandomState), 0.0f);
                    d_shapeList[i++] = new Sphere(center, centerTwo, 0.2f,
                        new lambertian(color4(RND * RND, RND * RND, RND * RND, 1.0f)));
                }
                else if (chosenMaterial < 0.95f)
                {
                    d_shapeList[i++] = new Sphere(center, 0.2f,
                        new metal(color4(0.5f * (1.0f * RND), 0.5f * (1.0f * RND), 0.5f * (1.0f * RND), 1.0f), 0.5f * RND));
                }
                else
                {
                    d_shapeList[i++] = new Sphere(center, 0.2f, new dielectric(1.5f));
                }
            }
        }

        d_shapeList[i++] = new Sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new dielectric(1.5f));

        d_shapeList[i++] = new Sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new lambertian(color4(0.4f, 0.2f, 0.1f, 1.0f)));
        
        d_shapeList[i++] = new Sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new metal(color4(0.7f, 0.6f, 0.5f, 1.0f), 0.0));

        bvhNode::prefillNodes(d_bvhTree, d_shapeList, objectCount);

        *d_camera = Camera(vec3(13.0f, 1.5f, -6.0f), vec3(0.0f, 1.0f, 0.0f),
            vec2(-12.0f, 155.0f), 30.0f, pX, pY, AAMethod::MSAA100,
            10.0f, 0.05f, color4::standardBackground());
    }
}
void randomSpheresScene::createScene(Shape**& d_shapeList, bvhNode*& h_bvhTree, bvhNode*& d_bvhTree, Camera*& d_camera, int pX, int pY, int randomSeed, int& listSize, int& treeSize)
{
    INIT_LIST_AND_TREE(objectCount);

    initializeRandomScene<<<1, 1>>>(d_shapeList, d_bvhTree, d_camera, pX, pY, objectCount, randomSeed);
}