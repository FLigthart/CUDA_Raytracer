#include "hip/hip_runtime.h"
#include "../../public/scenes/randomSpheresScene.h"

#include "hiprand/hiprand_kernel.h"
#include ""

#include "../../public/shapes/sphere.h"
#include "../../public/materials/dielectric.h"
#include "../../public/materials/metal.h"
#include "../../public/camera.h"

#define RND (hiprand_uniform(&localRandomState))

__global__ void InitializeScene(Shape** d_shapeList, Shape** d_world, Camera** d_camera, int pX, int pY, int objectCount, hiprandState* randomState)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        hiprandState localRandomState = *randomState;

        material* groundMaterial = new lambertian(color4(0.5f, 0.5f, 0.5f, 1.0f));

        d_shapeList[0] = new Sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f, groundMaterial);

        int i = 1;
        for (int a = -11; a < 11; a++)
        {
            for (int b = -11; b < 11; b++)
            {
                float chosenMaterial = RND;
                vec3 center(static_cast<float>(a) + RND * 0.5f, 0.2f, static_cast<float>(b) + RND * 0.5f);
                if (chosenMaterial < 0.8f)
                {
                    d_shapeList[i++] = new Sphere(center, 0.2f,
                        new lambertian(color4(RND * RND, RND * RND, RND * RND, 1.0f)));
                }
                else if (chosenMaterial < 0.95f)
                {
                    d_shapeList[i++] = new Sphere(center, 0.2f,
                        new metal(color4(0.5f * (1.0f * RND), 0.5f * (1.0f * RND), 0.5f * (1.0f * RND), 1.0f), 0.5f * RND));
                }
                else
                {
                    d_shapeList[i++] = new Sphere(center, 0.2f, new dielectric(1.5f));
                }
            }
        }

        d_shapeList[i++] = new Sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new dielectric(1.5f));

        d_shapeList[i++] = new Sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new lambertian(color4(0.4f, 0.2f, 0.1f, 1.0f)));
        
        d_shapeList[i++] = new Sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new metal(color4(0.7f, 0.6f, 0.5f, 1.0f), 0.0));

        *randomState = localRandomState;

        *d_world = new ShapeList(d_shapeList, objectCount);

        *d_camera = new Camera(vec3(13.0f, 1.5f, -6.0f), vec3(0.0f, 1.0f, 0.0f), vec2(-12.0f, 155.0f), 30.0f, pX, pY, AAMethod::MSAA1000, 10.0f, 0.05f); // standard camera
    }
}
void randomSpheresScene::CreateScene(Shape** d_shapeList, Shape** d_world, Camera** d_camera, int pX, int pY, hiprandState* randomState)
{
    InitializeScene<<<1, 1>>>(d_shapeList, d_world, d_camera, pX, pY, objectCount, randomState);
}