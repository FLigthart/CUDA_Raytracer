#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <time.h>

#include <hip/hip_runtime_api.h>
#include ""
#include "hip/hip_runtime.h"

#include "../public/structs/vec3.h"
#include "../public/ray.h"
#include "../public/camera.h"
#include "../public/shapes/sphere.h"
#include "../public/structs/color4.h"
#include "../public/exceptionChecker.h"
#include "../public/scenes.h"

__device__ vec3 calculateBackgroundColor(const Ray& r)
{
    vec3 normalizedDirection = r.direction().normalized();
    float t = (normalizedDirection.y() + 1.0f);
    return (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
}

__global__ void render(vec3* fb, int maxPixelX, int maxPixelY, vec3 lowerLeftCorner, vec3 horizontal, vec3 vertical, Scene* scene)
{
    int pixelStartX = threadIdx.x + blockIdx.x * blockDim.x;
    int pixelStartY = threadIdx.y + blockIdx.y * blockDim.y;

    if ((pixelStartX >= maxPixelX) || (pixelStartY >= maxPixelY)) return;   // Pixels that will be rendered are out of screen.

    int pixelIndex = pixelStartY * maxPixelX + pixelStartX; // Index of pixel in array.

    float u = static_cast<float>(pixelStartX) / static_cast<float>(maxPixelX);
    float v = static_cast<float>(pixelStartY) / static_cast<float>(maxPixelY);

    Ray r(scene->camera->position(), lowerLeftCorner + u * horizontal + v * vertical);

    HitInformation hitInformation;

    // ERROR IS HERE (LINE 54). SEE compute-sanitizer "D:\HomeProjects\CUDA_Raytracer\x64\Debug\CUDA_Raytracer.exe". 8 bytes read attempt. SHAPE IS INVALID

    Transform transform(vec3::one());
    if (scene->objectList && scene->objectList[0]->meshComponent->shape->checkIntersection(r, transform, hitInformation))
    {
        fb[pixelIndex] = scene->objectList[0]->meshComponent->color.getRGB();
    }
    else
    {
        fb[pixelIndex] = calculateBackgroundColor(r);
    }
}

int main() {

    int pX = 1920;
    int pY = 1080;

    float aspectRatioY = static_cast<float>(1080) / static_cast<float>(1920);

    float screenHeight = 2.0f;
    float screenWidth = screenHeight / aspectRatioY;
    float focalLength = 1.0f;

    // Divide threads into blocks to be sent to the gpu.
    int threadX = 12;
    int threadY = 12;

    std::cerr << "Rendering a " << pX << " x " << pY << " image " << "in " << threadX << " x " << threadY << " blocks.\n";

    int pixelCount = pX * pY;
    size_t memorySize = pixelCount * sizeof(vec3); // One vec3 (rgb) per pixel.

    vec3* fb;
    checkCudaErrors(hipMallocManaged(reinterpret_cast<void**>(&fb), memorySize));

    // Render a buffer
    dim3 blocks(pX / threadX + 1, pY / threadY + 1); //Block of one warp size.
    dim3 threads(threadX, threadY); // A block of amount of threads per block.

    Scene* d_scene = new Scene();

    simpleSphere(d_scene);

    // Ensure synchronization
    checkCudaErrors(hipDeviceSynchronize());

    render <<<blocks, threads>>> (fb, pX, pY,
        vec3(-screenWidth / 2, -screenHeight / 2, focalLength),
        vec3(screenWidth, 0.0, 0.0),
        vec3(0.0, screenHeight, 0.0), d_scene);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cerr << "Image rendered.\n";

    // Open the file
    std::ofstream ofs("image.ppm", std::ios::out | std::ios::trunc | std::ios::binary);  // Empty file before writing

    // Output an image
    ofs << "P3\n" << pX << " " << pY << "\n255\n";
    for (int y = pY - 1; y >= 0; y--)
    {
        for (int x = 0; x < pX; x++)
        {
            size_t pixelIndex = y * pX + x;

            float r = fb[pixelIndex].x();
            float g = fb[pixelIndex].y();
            float b = fb[pixelIndex].z();

            int ir = static_cast<int>(255.99 * r);
            int ig = static_cast<int>(255.99 * g);
            int ib = static_cast<int>(255.99 * b);

            ofs << ir << " " << ig << " " << ib << "\n";
        }
    }
    ofs.close();

    std::cerr << "Writing render to file finished.";

    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(d_scene));
}
