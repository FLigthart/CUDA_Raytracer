#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include <hip/hip_runtime_api.h>
#include ""
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"

#include "../public/structs/vec3.h"
#include "../public/ray.h"
#include "../public/camera.h"
#include "../public/shapes/sphere.h"
#include "../public/structs/color4.h"
#include "../public/exceptionChecker.h"

__device__ color4 calculateBackgroundColor(const Ray& r)
{
    vec3 normalizedDirection = r.direction().normalized();
    float t = (normalizedDirection.y() + 1.0f);
    vec3 rgb = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
    return color4(rgb.x(), rgb.y(), rgb.z(), 1.0f);
}

__global__ void createWorld(Shape** d_shapeList, Shape** d_world, Camera** d_camera, float screenHeight, float focalLength, float fov, int pX, int pY)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
        d_shapeList[0] = new Sphere(1.0f);
        d_shapeList[0]->transform.position = vec3(0.0f, 0.0f, 3.0f);

        d_shapeList[1] = new Sphere(2.0f);
        d_shapeList[1]->transform.position = vec3(2.5f, 0.0f, 4.0f);
        d_shapeList[1]->color = color4::green();

        *d_world = new ShapeList(d_shapeList, 2);

        *d_camera = new Camera(vec3(0.0f, 0.0f, -3.0f), vec3(0.0f, 1.0f, 0.0f), vec3(0.0f, 0.0f, 1.0f),
            screenHeight, focalLength, fov, pX, pY, AAMethod::None);
	}
}

__global__ void renderInitialize(int sX, int sY, hiprandState* randomState)  // Initializes random for every thread. Is used for MSAA.
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= sX) || (j >= sY)) return;

    int pixelIndex = j * sX + i;

    // Every thread gets the same seed, a different sequence number and no offset.
    hiprand_init(2024, pixelIndex, 0, &randomState[pixelIndex]);
}

__device__ vec3 renderNoAA(Shape** world, Camera** camera, int pixelStartX, int pixelStartY)
{
    HitInformation hitInformation;

    float u = static_cast<float>(pixelStartX) / static_cast<float>((*camera)->screenX);
    float v = static_cast<float>(pixelStartY) / static_cast<float>((*camera)->screenY);

    Ray r = (*camera)->makeRay(u, v);

    if ((*world)->checkIntersection(r, hitInformation))
    {
        return hitInformation.color.getRGB();
        //fb[pixelIndex] = 0.5f * vec3(hitInformation.normal.x() + 1.0f, hitInformation.normal.y() + 1.0f, hitInformation.normal.z() + 1.0f); //Display normals color
    }
    else
    {
        return calculateBackgroundColor(r).getRGB();
    }
}

__device__ vec3 renderAA(Shape** world, Camera** camera, int pixelStartX, int pixelStartY, hiprandState localRandomState, int sampleSize)
{
    color4 color = color4(0.0f, 0.0f, 0.0f, 1.0f);

    for (int i = 0; i < sampleSize; i++)
    {
        HitInformation hitInformation;

        float u = static_cast<float>(pixelStartX + hiprand_uniform(&localRandomState)) / static_cast<float>((*camera)->screenX);
        float v = static_cast<float>(pixelStartY + hiprand_uniform(&localRandomState)) / static_cast<float>((*camera)->screenY);

        Ray r = (*camera)->makeRay(u, v);

        if ((*world)->checkIntersection(r, hitInformation))
        {
            color += hitInformation.color;
            // color += 0.5f * vec3(hitInformation.normal.x() + 1.0f, hitInformation.normal.y() + 1.0f, hitInformation.normal.z() + 1.0f); //Display normals color
        }
        else
        {
            color += calculateBackgroundColor(r);
        }
    }

    return (color.getRGB() / static_cast<float>(sampleSize));
}

__global__ void render(vec3* fb, Camera** camera, Shape** world, hiprandState* randomState)
{
    int pixelStartX = threadIdx.x + blockIdx.x * blockDim.x;
    int pixelStartY = threadIdx.y + blockIdx.y * blockDim.y;

    if ((pixelStartX >= (*camera)->screenX) || (pixelStartY >= (*camera)->screenY)) return;   // Pixels that will be rendered are out of screen.

    int pixelIndex = pixelStartY * (*camera)->screenX + pixelStartX; // Index of pixel in array.

    hiprandState localRandomState = randomState[pixelIndex];

    switch ((*camera)->aaMethod)
    {
	    case MSAA4:
            fb[pixelIndex] = renderAA(world, camera, pixelStartX, pixelStartY, localRandomState, 4);
            break;

	    case MSAA8:
            fb[pixelIndex] = renderAA(world, camera, pixelStartX, pixelStartY, localRandomState, 8);
            break;

	    case MSAA16:
            fb[pixelIndex] = renderAA(world, camera, pixelStartX, pixelStartY, localRandomState, 16);
            break;

	    default: // No AA/Non-added methods
            fb[pixelIndex] = renderNoAA(world, camera, pixelStartX, pixelStartY);
    }
}

__global__ void freeWorld(Shape** shapeList, Shape** world, Camera** camera)
{
	for (int i = 0; i < 2; i++)
	{
        delete shapeList[i];
	}

    delete *world;
    delete *camera;
}

int main()
{
    int pX = 400; //1920
    int pY = 235; //1080

    float screenHeight = 2.0f;
    float focalLength = 1.0f;
    float fov = 20.0f; //No effect yet

    // Divide threads into blocks to be sent to the gpu.
    int threadX = 12;
    int threadY = 12;

    std::cerr << "Rendering a " << pX << " x " << pY << " image " << "in " << threadX << " x " << threadY << " blocks.\n";

    vec3* fb;

    int pixelCount = pX * pY;
    size_t memorySize = pixelCount * sizeof(vec3); // One vec3 (rgb) per pixel.
    checkCudaErrors(hipMallocManaged(reinterpret_cast<void**>(&fb), memorySize));

    Camera** d_camera;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_camera), sizeof(Camera*)));

    int shapeListSize = 2;
    Shape** d_shapeList;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_shapeList), shapeListSize * sizeof(Shape*)));

    Shape** d_world;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_world), sizeof(Shape*)));

    createWorld<<<1, 1>>>(d_shapeList, d_world, d_camera, screenHeight, focalLength, fov, pX, pY);

    // Render a buffer
    dim3 blocks(pX / threadX + 1, pY / threadY + 1); //Block of one warp size.
    dim3 threads(threadX, threadY); // A block of amount of threads per block.

    // Allocate random state
    hiprandState* d_randomState;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_randomState), pixelCount * sizeof(hiprandState)));

    // Ensure synchronization
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    renderInitialize<<<blocks, threads>>>(pX, pY, d_randomState);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(fb, d_camera, d_world, d_randomState);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cerr << "Image rendered.\n";

    // Open the file
    std::ofstream ofs("image.ppm", std::ios::out | std::ios::trunc | std::ios::binary);  // Empty file before writing


    // Output an image
    ofs << "P3\n" << pX << " " << pY << "\n255\n";
    for (int y = pY - 1; y >= 0; y--)
    {
        for (int x = 0; x < pX; x++)
        {
            size_t pixelIndex = y * pX + x;

            float r = fb[pixelIndex].x();
            float g = fb[pixelIndex].y();
            float b = fb[pixelIndex].z();

            int ir = static_cast<int>(255.99 * r);
            int ig = static_cast<int>(255.99 * g);
            int ib = static_cast<int>(255.99 * b);

            ofs << ir << " " << ig << " " << ib << "\n";
        }
    }
    ofs.close();

    std::cerr << "Writing render to file finished.";

    checkCudaErrors(hipDeviceSynchronize());
    freeWorld<<<1, 1 >>>(d_shapeList, d_world, d_camera);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_shapeList));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_randomState));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}
